#include "hip/hip_runtime.h"
#include"testRNG.h"
#include<iostream>
#include<hiprand/hiprand_kernel.h>
#include"helper_timer.h"

#include<trng/yarn5s.hpp>
#include<trng/normal_dist.hpp>

__global__ void initRNG(hiprandState *const rngStates,const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}
__global__ void initRNG_Philox(hiprandStatePhilox4_32_10_t *const rngStates,
					    const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}

__global__ void initRNG_MRG(hiprandStateMRG32k3a *const rngStates,
					    const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}

__global__ void generator(hiprandState *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	unsigned int timeStep=1000;

	hiprandState localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<timeStep;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}
__global__ void generatorP(hiprandStatePhilox4_32_10_t *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	unsigned int timeStep=1000;

	hiprandStatePhilox4_32_10_t localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<timeStep;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}
__global__ void generatorM(hiprandStateMRG32k3a *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	unsigned int timeStep=1000;

	hiprandStateMRG32k3a localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<timeStep;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}

void generator_yarn5s(trng::yarn5s rngState,const unsigned int numSims,const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	unsigned int timeStep=1000;

	trng::normal_dist<float> n;
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<timeStep;t++)
		{
			double x=n(rngState);
		}
	}
	return;
}

template<typename Real>
testRNG<Real>::testRNG(unsigned int numSims, 
					   unsigned int numTimeSteps,
					   unsigned int threadBlockSize,
					   unsigned int seed
					  )
					 :m_numSims(numSims),
					  m_numTimeSteps(numTimeSteps),
					  m_threadBlockSize(threadBlockSize),
					  m_seed(seed)
{
	return ;
}

template <typename Real>
void testRNG<Real>::operator()()
{
	hipError_t cudaResult=hipSuccess; //cuda函数运行结果标识
	hiprandState *d_rngStates=(hiprandState*)malloc(sizeof(hiprandState)*m_numSims);
	hiprandStatePhilox4_32_10_t *d_rngStatesP=(hiprandStatePhilox4_32_10_t*)malloc(sizeof(hiprandStatePhilox4_32_10_t)*m_numSims);
	hiprandStateMRG32k3a *d_rngStatesM=(hiprandStateMRG32k3a*)malloc(sizeof(hiprandStateMRG32k3a)*m_numSims);

	dim3 block,grid;
	block.x=m_threadBlockSize;
	block.y=1;
	block.z=1;

	grid.x=m_numSims/m_threadBlockSize;
	grid.y=1;
	grid.z=1;

	cudaResult=hipMalloc((void **)&d_rngStates,m_numSims*sizeof(hiprandState));
	cudaResult=hipMalloc((void **)&d_rngStatesP,m_numSims*sizeof(hiprandStatePhilox4_32_10_t));
	cudaResult=hipMalloc((void **)&d_rngStatesM,m_numSims*sizeof(hiprandStateMRG32k3a));
	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

	initRNG<<<grid,block>>>(d_rngStates,m_seed);
	generator<<<grid,block>>>(d_rngStates,m_numSims,m_numTimeSteps);

	initRNG_Philox<<<grid,block>>>(d_rngStatesP,m_seed);
	generatorP<<<grid,block>>>(d_rngStatesP,m_numSims,m_numTimeSteps);

	initRNG_MRG<<<grid,block>>>(d_rngStatesM,m_seed);
	generatorM<<<grid,block>>>(d_rngStatesM,m_numSims,m_numTimeSteps);

	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

	hipFree(d_rngStates);
	hipFree(d_rngStatesP);
	hipFree(d_rngStatesM);


	/******************************************************************/
	trng::yarn5s rngState;
	generator_yarn5s<<<grid,block>>>(rngState,m_numSims,m_numTimeSteps);


	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

}
template class testRNG<float>;
template class testRNG<double>;
