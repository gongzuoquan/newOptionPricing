#include "hip/hip_runtime.h"
#include"testRNG.h"
#include<iostream>
#include<hiprand/hiprand_kernel.h>
#include"helper_timer.h"
#include<trng/yarn5s.hpp>
#include<trng/yarn2.hpp>
#include<trng/normal_dist.hpp>

__global__ void initRNG(hiprandState *const rngStates,const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}
__global__ void initRNG(hiprandStatePhilox4_32_10_t *const rngStates,
					    const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}

__global__ void initRNG(hiprandStateMRG32k3a *const rngStates,
					    const unsigned int seed)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x;
	hiprand_init(seed,tid,0,&rngStates[tid]);
	return ;
}

__global__ void generator(hiprandState *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目

	hiprandState localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<numTimeSteps;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}
__global__ void generator(hiprandStatePhilox4_32_10_t *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目

	hiprandStatePhilox4_32_10_t localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<numTimeSteps;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}
__global__ void generator(hiprandStateMRG32k3a *const  rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目

	hiprandStateMRG32k3a localState=rngStates[tid];
	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<numTimeSteps;t++)
		{
			hiprand_normal(&localState);
		}
	}
	return ;
}

__global__ void generator(trng::yarn5s rngState,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	rngState.jump(tid);
	trng::normal_dist<double> trng_normal(0.0,1.0);

	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<numTimeSteps;t++)
		{
			trng_normal(rngState);
		}
	}
	return ;
}

__global__ void generatorLeap(trng::yarn2 *rngStates,
						  const unsigned int numSims,
						  const unsigned int numTimeSteps)
{
	unsigned int tid=blockIdx.x*blockDim.x+threadIdx.x; //当前thread在全局的索引号
	unsigned int step=gridDim.x*blockDim.x; //当前应用程序中全部thread数目
	trng::normal_dist<double> trng_normal(0.0,1.0);

	for(unsigned int i=tid;i<numSims;i+=step)
	{
		for(unsigned int t=0;t<numTimeSteps;t++)
		{
			trng_normal(rngStates[tid]);
		}
	}
	return ;
}

template<typename Real>
testRNG<Real>::testRNG(unsigned int numSims, 
					   unsigned int numTimeSteps,
					   unsigned int threadBlockSize,
					   unsigned int seed
					  )
					 :m_numSims(numSims),
					  m_numTimeSteps(numTimeSteps),
					  m_threadBlockSize(threadBlockSize),
					  m_seed(seed)
{
	return ;
}

template <typename Real>
void testRNG<Real>::operator()()
{
	hipError_t cudaResult=hipSuccess; //cuda函数运行结果标识
	hiprandState *d_rngStates;
	//hiprandState *d_rngStates=(hiprandState*)malloc(sizeof(hiprandState)*m_numSims);
	hiprandStatePhilox4_32_10_t *d_rngStatesP=(hiprandStatePhilox4_32_10_t*)malloc(sizeof(hiprandStatePhilox4_32_10_t)*m_numSims);
	hiprandStateMRG32k3a *d_rngStatesM=(hiprandStateMRG32k3a*)malloc(sizeof(hiprandStateMRG32k3a)*m_numSims);

	dim3 block,grid;
	block.x=m_threadBlockSize;
	block.y=1;
	block.z=1;

	grid.x=m_numSims/m_threadBlockSize;
	grid.y=1;
	grid.z=1;

	cudaResult=hipMalloc((void **)&d_rngStates,m_numSims*sizeof(hiprandState));
	cudaResult=hipMalloc((void **)&d_rngStatesP,m_numSims*sizeof(hiprandStatePhilox4_32_10_t));
	cudaResult=hipMalloc((void **)&d_rngStatesM,m_numSims*sizeof(hiprandStateMRG32k3a));
	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

	initRNG<<<grid,block>>>(d_rngStates,m_seed);
	generator<<<grid,block>>>(d_rngStates,m_numSims,m_numTimeSteps);


	initRNG<<<grid,block>>>(d_rngStatesP,m_seed);
	generator<<<grid,block>>>(d_rngStatesP,m_numSims,m_numTimeSteps);


	initRNG<<<grid,block>>>(d_rngStatesM,m_seed);
	generator<<<grid,block>>>(d_rngStatesM,m_numSims,m_numTimeSteps);


	trng::yarn5s d_rngStateT;
	generator<<<grid,block>>>(d_rngStateT,m_numSims,m_numTimeSteps);


	hipDeviceSynchronize();

	//StopWatchInterface *timer=NULL;
	//sdkCreateTimer(&timer);

	//sdkStartTimer(&timer);
	trng::yarn2 *rx=new trng::yarn2[m_numSims];
	for(int i=0;i<m_numSims;i++)
	{
		rx[i].split(m_numSims,i);
	}
	//sdkStopTimer(&timer);

	trng::yarn2 *rx_device;
	hipMalloc(&rx_device,m_numSims*sizeof(*rx_device));
	hipMemcpy(rx_device, rx, m_numSims*sizeof(*rx),hipMemcpyHostToDevice);
	generatorLeap<<<grid,block>>>(rx_device,m_numSims,m_numTimeSteps);

	//float elapsedTime=sdkGetAverageTimerValue(&timer)/1000.0f;
	//std::cout<<"        run time : "<<elapsedTime<<std::endl;
	//sdkDeleteTimer(&timer);
	//timer=NULL;


	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

	hipFree(d_rngStates);
	hipFree(d_rngStatesP);
	hipFree(d_rngStatesM);


	//std::cout<<"ok"<<std::endl;
	hipDeviceSynchronize();

}
template class testRNG<float>;
template class testRNG<double>;
